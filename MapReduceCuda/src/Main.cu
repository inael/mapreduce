
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
int maxWords = 13;
char *words[13][2];
int qtyWord = 0;

int map(char *name, char *text);
int digit_to_int(char d);
void emit(char *word, char *count);
void reduce(char * key, char * valuesByKey);
struct token {
	char word;
	char qtd;
};

int main() {
	printf("Inicio \n");
	char *name = "frutas";
	char text[] = "PERA-MA�A BANANA UVA-MELANCIA LARANJA PERA-UVA MORANDO LARANJA ACEROLA CAQUI UVA";
	map(name, text);

	return 0;
}

int map(char* name, char text[]) {
	printf("Map \n");
	printf("%s\n",text);
	char s[2] = "-";
	char *token;
	printf("antes do strtok");
	/* get the first token */

	token = strtok(text, s);
	printf("Primeiro %s\n",token);
	/* walk through other tokens */
	while (token) {
		printf(" %s\n", token);
		token = strtok(NULL, s);
	}

	return 0;
}

int digit_to_int(char d) {
	char str[2];
	str[0] = d;
	str[1] = '\0';
	return (int) strtol(str, NULL, 10);
}
void emit(char *word, char *count) {
	if (qtyWord <= 10) {
		words[qtyWord][0] = word;
		words[qtyWord][1] = count;
		qtyWord++;
	}
	printf("%s %u\n", word, count);
}

void reduce(char * key, char * valuesByKey) {
	int sum = 0;
	int i =0;
	for (i = 0; i < sizeof(valuesByKey) / sizeof(int); i++) {
		sum = +digit_to_int(valuesByKey[i]);
	}
	printf("%s %u\n", key, sum);

}

/*
 * function map(String name, String document):
 // name: document name
 // document: document contents
 for each word w in document:
 emit (w, 1)

 function reduce(String word, Iterator partialCounts):
 // word: a word
 // partialCounts: a list of aggregated partial counts
 sum = 0
 for each pc in partialCounts:
 sum += ParseInt(pc)
 emit (word, sum)
 *
 */

